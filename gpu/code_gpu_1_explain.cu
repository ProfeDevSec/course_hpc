#include "hip/hip_runtime.h"
/************************************************************************************
 * Multiplicación de matrices cuadradas NxN en CUDA (versión didáctica)
 *
 * Objetivo:
 *  - Mostrar el flujo completo: reservar memoria en host y device, copiar datos,
 *    configurar rejilla/bloques, lanzar kernel, sincronizar, copiar resultados y validar.
 *  - Incluir medición de tiempo GPU (cudaEvent) y comparación con una versión CPU.
 *
 * Requisitos:
 *  - Compilar con nvcc (CUDA Toolkit). En Windows, asegúrate de tener MSVC instalado
 *    y compila desde "x64 Native Tools Command Prompt for VS".
 *
 * Compilación:
 *  nvcc -O2 -o matrixMul main.cu
 *
 * Ejecución:
 *  ./matrixMul
 ************************************************************************************/

#include <stdio.h>                  // printf, fprintf
#include <stdlib.h>                 // malloc, free, rand, srand
#include <math.h>                   // fabsf
#include <hip/hip_runtime.h>           // API de CUDA Runtime
#include <> // Parámetros de lanzamiento (MSVC/Windows)

#ifndef N
#define N 1024   // Tamaño de matriz: N x N (puedes cambiarlo; debe ser múltiplo o no de 16, da igual)
#endif

/*********** Utilidad: macro para chequear errores de llamadas CUDA ***********/
#define CUDA_CHECK(call) do {                                                     \
    hipError_t _e = (call);                                                      \
    if (_e != hipSuccess) {                                                      \
        fprintf(stderr, "CUDA error en %s:%d: %s\n", __FILE__, __LINE__,          \
                hipGetErrorString(_e));                                          \
        exit(EXIT_FAILURE);                                                       \
    }                                                                             \
} while (0)

/*********** Kernel de CUDA: C = A * B (matrices cuadradas NxN) ****************
 * Cada hilo calcula un elemento C[row, col].
 * Índices 2D:
 *   - blockIdx.x, blockIdx.y: coordenadas del bloque
 *   - threadIdx.x, threadIdx.y: coordenadas del hilo dentro del bloque
 *   - blockDim.x, blockDim.y: tamaño del bloque (hilos por dimensión)
 *******************************************************************************/
__global__ void matrixMultiplyGPU(const float *A, const float *B, float *C, int n) {
    // Calcular la fila y columna globales que le tocan a este hilo
    int row = blockIdx.y * blockDim.y + threadIdx.y;  // eje Y -> filas
    int col = blockIdx.x * blockDim.x + threadIdx.x;  // eje X -> columnas

    // Validación de límites (la rejilla puede cubrir "de más")
    if (row >= n || col >= n) return;

    // Acumulador para el producto punto de la fila 'row' de A y la columna 'col' de B
    float sum = 0.0f;

    // Bucle interno: recorrer k para A[row, k] * B[k, col]
    for (int k = 0; k < n; ++k) {
        sum += A[row * n + k] * B[k * n + col];
    }

    // Escribir el resultado en C
    C[row * n + col] = sum;
}

/*********** Inicialización de matrices en host ********************************
 * Rellena con enteros aleatorios [0..99] convertidos a float.
 *******************************************************************************/
void initializeMatrix(float *m, int elements) {
    for (int i = 0; i < elements; ++i) {
        m[i] = (float)(rand() % 100);
    }
}

/*********** Versión CPU de referencia *****************************************
 * Implementación sencilla O(N^3) para verificar la corrección del resultado GPU.
 *******************************************************************************/
void matrixMultiplyCPU(const float *A, const float *B, float *C, int n) {
    for (int i = 0; i < n; ++i) {          // filas de A
        for (int j = 0; j < n; ++j) {      // columnas de B
            float sum = 0.0f;
            for (int k = 0; k < n; ++k) {  // producto punto
                sum += A[i * n + k] * B[k * n + j];
            }
            C[i * n + j] = sum;
        }
    }
}

/*********** Comparación numérica **********************************************
 * Calcula el error máximo absoluto entre C_gpu y C_cpu.
 * Devuelve true si el error es aceptable (<= tol).
 *******************************************************************************/
bool compareMatrices(const float *A, const float *B, int elements, float tol,
                     float *out_max_abs_err) {
    float max_err = 0.0f;
    for (int i = 0; i < elements; ++i) {
        float err = fabsf(A[i] - B[i]);
        if (err > max_err) max_err = err;
        if (err > tol) { // Early exit posible si quieres
            // sigue para medir el máximo real
        }
    }
    if (out_max_abs_err) *out_max_abs_err = max_err;
    return max_err <= tol;
}

/*********** (Opcional) Imprimir submatriz pequeña *****************************/
void printMatrixBlock(const float *M, int n, int rows=8, int cols=8) {
    int rmax = (rows < n) ? rows : n;
    int cmax = (cols < n) ? cols : n;
    for (int i = 0; i < rmax; ++i) {
        for (int j = 0; j < cmax; ++j) {
            printf("%7.2f ", M[i * n + j]);
        }
        printf("\n");
    }
}

int main() {
    /***** 1) Preparación de datos en Host *****/
    srand(1234); // Semilla fija para reproducibilidad

    const int n = N;                                        // tamaño lógico
    const size_t elements = (size_t)n * (size_t)n;          // número de floats por matriz
    const size_t bytes    = elements * sizeof(float);       // tamaño en bytes por matriz

    // Reservar matrices en Host (CPU)
    float *h_A  = (float*)malloc(bytes);
    float *h_B  = (float*)malloc(bytes);
    float *h_Cg = (float*)malloc(bytes); // resultado desde GPU
    float *h_Cc = (float*)malloc(bytes); // resultado CPU (para validar)
    if (!h_A || !h_B || !h_Cg || !h_Cc) {
        fprintf(stderr, "Error: no se pudo reservar memoria en host.\n");
        return EXIT_FAILURE;
    }

    // Inicializar A y B con valores aleatorios
    initializeMatrix(h_A, (int)elements);
    initializeMatrix(h_B, (int)elements);

    /***** 2) Reservar memoria en Device (GPU) *****/
    float *d_A = nullptr, *d_B = nullptr, *d_C = nullptr;
    CUDA_CHECK(hipMalloc((void**)&d_A, bytes));
    CUDA_CHECK(hipMalloc((void**)&d_B, bytes));
    CUDA_CHECK(hipMalloc((void**)&d_C, bytes));

    /***** 3) Copiar A y B del Host al Device *****/
    CUDA_CHECK(hipMemcpy(d_A, h_A, bytes, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_B, h_B, bytes, hipMemcpyHostToDevice));

    /***** 4) Configuración de ejecución ****************************************
     * Elegimos bloques 16x16 (=256 hilos por bloque), bastante estándar.
     * Calculamos cuántos bloques se necesitan para cubrir N en cada dimensión:
     *   grid.x = ceil(N / blockDim.x)
     *   grid.y = ceil(N / blockDim.y)
     ****************************************************************************/
    dim3 block(16, 16);
    dim3 grid((n + block.x - 1) / block.x,
              (n + block.y - 1) / block.y);

    /***** 5) Medición de tiempo GPU con hipEvent_t ****************************/
    hipEvent_t start, stop;
    CUDA_CHECK(hipEventCreate(&start));
    CUDA_CHECK(hipEventCreate(&stop));

    // Registrar tiempo antes del kernel
    CUDA_CHECK(hipEventRecord(start));

    /***** 6) Lanzamiento del kernel ********************************************/
    matrixMultiplyGPU<<<grid, block>>>(d_A, d_B, d_C, n);

    // Comprobar si el lanzamiento generó algún error asíncrono
    CUDA_CHECK(hipGetLastError());

    // Sincronizar para que el kernel termine antes de medir tiempo y continuar
    CUDA_CHECK(hipDeviceSynchronize());

    // Registrar tiempo después del kernel y calcular milisegundos
    CUDA_CHECK(hipEventRecord(stop));
    CUDA_CHECK(hipEventSynchronize(stop));
    float msGPU = 0.0f;
    CUDA_CHECK(hipEventElapsedTime(&msGPU, start, stop));

    // Destruir eventos
    CUDA_CHECK(hipEventDestroy(start));
    CUDA_CHECK(hipEventDestroy(stop));

    /***** 7) Copiar resultado de vuelta al Host *****/
    CUDA_CHECK(hipMemcpy(h_Cg, d_C, bytes, hipMemcpyDeviceToHost));

    /***** 8) (Opcional) Calcular resultado en CPU para validar *****************/
    // Ojo: CPU O(N^3) puede tardar para N grandes. Con N=1024 es costoso.
    // Si quieres validar rápido, prueba con N=256 o N=512.
    printf("Calculando referencia CPU (puede tardar con N=%d)...\n", n);
    float msCPU = 0.0f;
    {
        // Cronómetro simple con eventos de CPU (poco preciso). Si deseas
        // medir mejor en Windows, usa QueryPerformanceCounter.
        clock_t c0 = clock();
        matrixMultiplyCPU(h_A, h_B, h_Cc, n);
        clock_t c1 = clock();
        msCPU = 1000.0f * (float)(c1 - c0) / (float)CLOCKS_PER_SEC;
    }

    /***** 9) Comparar resultados ***********************************************/
    float max_abs_err = 0.0f;
    const float tol = 1e-3f; // tolerancia típica para floats con sumas acumuladas
    bool ok = compareMatrices(h_Cg, h_Cc, (int)elements, tol, &max_abs_err);

    /***** 10) Reporte **********************************************************/
    printf("\n=== Reporte ===\n");
    printf("N = %d\n", n);
    printf("Tiempo GPU (kernel) : %.3f ms\n", msGPU);
    printf("Tiempo CPU (ref)    : %.3f ms\n", msCPU);
    printf("Error max abs       : %.6f\n", max_abs_err);
    printf("Validación          : %s\n", ok ? "OK (dentro de tolerancia)" : "FALLO (excede tolerancia)");

    // (Opcional) imprime un bloque 8x8 de la salida para inspección visual
    // printf("\nBloque 8x8 de C (GPU):\n");
    // printMatrixBlock(h_Cg, n, 8, 8);

    /***** 11) Liberar recursos *************************************************/
    CUDA_CHECK(hipFree(d_A));
    CUDA_CHECK(hipFree(d_B));
    CUDA_CHECK(hipFree(d_C));
    free(h_A);
    free(h_B);
    free(h_Cg);
    free(h_Cc);

    return ok ? EXIT_SUCCESS : EXIT_FAILURE;
}
