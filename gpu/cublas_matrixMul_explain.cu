/************************************************************************************
 * SGEMM con cuBLAS (A * B = C) usando datos en ROW-MAJOR (C/C++) con "truco" de transposición
 *
 * Propósito
 * ---------
 * - Mostrar una implementación mínima, segura y rápida de multiplicación de matrices
 *   de punto flotante simple (SGEMM) en GPU usando cuBLAS.
 * - El código de aplicación trabaja en orden de memoria ROW-MAJOR (estilo C/C++).
 *   cuBLAS, por defecto, interpreta los punteros en COLUMN-MAJOR (Fortran).
 *   Para mantener nuestros datos en row-major sin reordenarlos, usamos el "truco":
 *      C_row = A_row * B_row   equivale a   C_col = (B_col)^T * (A_col)^T
 *   Es decir, pedimos a cuBLAS que multiplique B^T por A^T (ambas transpuestas),
 *   lo que produce el mismo resultado que A*B en convención row-major.
 *
 * Pasos principales
 * -----------------
 * 1) Reservar y llenar matrices en host (A, B) [row-major].
 * 2) Reservar matrices en device (d_A, d_B, d_C).
 * 3) Copiar A y B a la GPU.
 * 4) Crear handle cuBLAS y (opcional) configurar math mode.
 * 5) Llamar a cublasSgemm con (op_T, op_T) y orden B primero, luego A.
 * 6) Copiar C de vuelta a host.
 * 7) Liberar recursos (device/host) y destruir handle.
 *
 * Notas de rendimiento
 * --------------------
 * - Usamos memoria "page-locked" (cudaMallocHost) para acelerar cudaMemcpy.
 * - Compilar con -O3 y enlazar con -lcublas.
 * - Para FP32 en GPUs Ampere/Hopper puedes activar Tensor Cores (TF32):
 *     cublasSetMathMode(handle, CUBLAS_TF32_TENSOR_OP_MATH);
 *   (menor precisión, mayor rendimiento).
 *
 * Compilación (Windows/Linux)
 * ---------------------------
 *   nvcc -O3 -o sgemm main.cu -lcublas
 *
 * Validación
 * ----------
 * - Si necesitas validar, compara contra una implementación CPU para tamaños pequeños
 *   (p.ej., N=256), o usa una segunda llamada cuBLAS con layouts alternativos.
 ************************************************************************************/

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>

#ifndef N
#define N 1024   // Tamaño por defecto: matrices cuadradas N x N
#endif

/********** Macros de comprobación de errores (CUDA y cuBLAS) ******************/
#define CUDA_CHECK(x) do { \
    hipError_t e = (x); \
    if (e != hipSuccess) { \
        fprintf(stderr, "CUDA %s:%d: %s\n", __FILE__, __LINE__, hipGetErrorString(e)); \
        exit(EXIT_FAILURE); \
    } \
} while(0)

#define CUBLAS_CHECK(x) do { \
    hipblasStatus_t s = (x); \
    if (s != HIPBLAS_STATUS_SUCCESS) { \
        fprintf(stderr, "cuBLAS %s:%d: status=%d\n", __FILE__, __LINE__, (int)s); \
        exit(EXIT_FAILURE); \
    } \
} while(0)

/**
 * initializeMatrix:
 * Llena un buffer row-major de 'elements' floats con valores pseudoaleatorios [0..99].
 * Usamos srand(1234) en main() para reproducibilidad.
 */
static void initializeMatrix(float *m, size_t elements) {
    for (size_t i = 0; i < elements; ++i) m[i] = (float)(rand() % 100);
}

int main() {
    /***** Semilla fija para que las ejecuciones sean reproducibles *****/
    srand(1234);

    /***** Parámetros y tamaños *****/
    const int n = N;                                 // Dimensión lógica
    const size_t elems = (size_t)n * (size_t)n;      // Número de elementos por matriz
    const size_t bytes = elems * sizeof(float);      // Tamaño en bytes por matriz

    /***** Reservas en host (page-locked para copias más rápidas) ********
     * Nota: Si prefieres no usar page-locked, sustituye cudaMallocHost por malloc
     * y CUDA_CHECK(cudaFreeHost(...)) por free(...).
     *********************************************************************/
    float *h_A = nullptr, *h_B = nullptr, *h_C = nullptr;
    CUDA_CHECK(hipHostMalloc(&h_A, bytes, hipHostMallocDefault));
    CUDA_CHECK(hipHostMalloc(&h_B, bytes, hipHostMallocDefault));
    CUDA_CHECK(hipHostMalloc(&h_C, bytes, hipHostMallocDefault));

    /***** Inicialización de A y B en el host (row-major) ****************/
    initializeMatrix(h_A, elems);
    initializeMatrix(h_B, elems);

    /***** Reservas en device ********************************************/
    float *d_A = nullptr, *d_B = nullptr, *d_C = nullptr;
    CUDA_CHECK(hipMalloc(&d_A, bytes));
    CUDA_CHECK(hipMalloc(&d_B, bytes));
    CUDA_CHECK(hipMalloc(&d_C, bytes));

    /***** Copias Host -> Device *****************************************/
    CUDA_CHECK(hipMemcpy(d_A, h_A, bytes, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_B, h_B, bytes, hipMemcpyHostToDevice));

    /***** Crear handle cuBLAS *******************************************/
    hipblasHandle_t handle;
    CUBLAS_CHECK(hipblasCreate(&handle));

    // (Opcional) Activa Tensor Cores en FP32 (TF32). Úsalo si aceptas menor precisión:
    // CUBLAS_CHECK(cublasSetMathMode(handle, CUBLAS_TF32_TENSOR_OP_MATH));

    /***** Parámetros escalares SGEMM: C = alpha * (opA(A) * opB(B)) + beta * C *****/
    const float alpha = 1.0f;
    const float beta  = 0.0f;

    /***** SGEMM con "truco" para conservar ROW-MAJOR en la app **********
     *
     * Queremos: C_row = A_row * B_row  (datos en row-major)
     * cuBLAS asume COLUMN-MAJOR. La identidad útil es:
     *   C_row = A_row * B_row   <=>   C_col = (B_col)^T * (A_col)^T
     * Esto se implementa llamando a SGEMM con:
     *   op(A) = Transpuesta,  op(B) = Transpuesta,  y pasando primero B y luego A.
     *
     * Firma de cublasSgemm (col-major):
     *   cublasSgemm(handle, opA, opB,
     *               m, n, k,
     *               &alpha,
     *               A, lda,
     *               B, ldb,
     *               &beta,
     *               C, ldc);
     *
     * Para matrices cuadradas N:
     *   m = n = k = N
     *   lda = ldb = ldc = N (leading dimension)
     *********************************************************************/
    CUBLAS_CHECK(hipblasSgemm(
        handle,
        HIPBLAS_OP_T, HIPBLAS_OP_T,   // transponer ambos operandos
        n, n, n,                    // m, n, k
        &alpha,
        d_B, n,                     // B primero (será B^T internamente)
        d_A, n,                     // luego A (será A^T internamente)
        &beta,
        d_C, n                      // salida C (col-major visto por cuBLAS; row-major para nosotros)
    ));

    /***** Copia Device -> Host ******************************************/
    CUDA_CHECK(hipMemcpy(h_C, d_C, bytes, hipMemcpyDeviceToHost));

    /***** Limpieza de recursos ******************************************/
    CUBLAS_CHECK(hipblasDestroy(handle));
    CUDA_CHECK(hipFree(d_A));
    CUDA_CHECK(hipFree(d_B));
    CUDA_CHECK(hipFree(d_C));
    CUDA_CHECK(hipHostFree(h_A));
    CUDA_CHECK(hipHostFree(h_B));
    CUDA_CHECK(hipHostFree(h_C));

    /***** Fin ***********************************************************/
    printf("SGEMM con cuBLAS completado (row-major vía transposición).\n");
    return 0;
}
