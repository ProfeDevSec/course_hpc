#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>


#define N 1024  // Matrices NxN

#define CUDA_CHECK(call) do {                                      \
    hipError_t _e = (call);                                       \
    if (_e != hipSuccess) {                                       \
        fprintf(stderr, "CUDA error %s:%d: %s\n",                  \
                __FILE__, __LINE__, hipGetErrorString(_e));       \
        exit(EXIT_FAILURE);                                        \
    }                                                              \
} while (0)

__global__ void matrixMultiplyGPU(const float *A, const float *B, float *C, int n) {
    int row = blockIdx.y * blockDim.y + threadIdx.y; // y -> filas
    int col = blockIdx.x * blockDim.x + threadIdx.x; // x -> columnas
    if (row >= n || col >= n) return;

    float sum = 0.0f;
    for (int i = 0; i < n; i++) {
        sum += A[row * n + i] * B[i * n + col];
    }
    C[row * n + col] = sum;
}

void initializeMatrix(float *matrix, int elements) {
    for (int i = 0; i < elements; i++) {
        matrix[i] = (float)(rand() % 100);
    }
}

void printMatrix(const float *matrix, int n) {
    for (int i = 0; i < n; i++) {
        for (int j = 0; j < n; j++) {
            printf("%.2f ", matrix[i * n + j]);
        }
        printf("\n");
    }
}

int main() {
    srand(1234);

    const size_t bytes = (size_t)N * (size_t)N * sizeof(float);

    // Host
    float *h_A = (float*)malloc(bytes);
    float *h_B = (float*)malloc(bytes);
    float *h_C = (float*)malloc(bytes);
    if (!h_A || !h_B || !h_C) {
        fprintf(stderr, "No se pudo asignar memoria en host.\n");
        return EXIT_FAILURE;
    }

    initializeMatrix(h_A, N * N);
    initializeMatrix(h_B, N * N);

    // Device
    float *d_A = nullptr, *d_B = nullptr, *d_C = nullptr;
    CUDA_CHECK(hipMalloc((void**)&d_A, bytes));
    CUDA_CHECK(hipMalloc((void**)&d_B, bytes));
    CUDA_CHECK(hipMalloc((void**)&d_C, bytes));

    CUDA_CHECK(hipMemcpy(d_A, h_A, bytes, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_B, h_B, bytes, hipMemcpyHostToDevice));

    // Configuración de ejecución
    dim3 threadsPerBlock(16, 16);
    dim3 blocksPerGrid((N + threadsPerBlock.x - 1) / threadsPerBlock.x,
                       (N + threadsPerBlock.y - 1) / threadsPerBlock.y);

    // ¡Lanzamiento correcto del kernel!
    matrixMultiplyGPU<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, N);
    CUDA_CHECK(hipGetLastError());
    CUDA_CHECK(hipDeviceSynchronize());

    // Copiar resultado de vuelta
    CUDA_CHECK(hipMemcpy(h_C, d_C, bytes, hipMemcpyDeviceToHost));

    // (Opcional) imprimir un subbloque pequeño para verificar
    printMatrix(h_C, 8);

    // Liberar
    CUDA_CHECK(hipFree(d_A));
    CUDA_CHECK(hipFree(d_B));
    CUDA_CHECK(hipFree(d_C));
    free(h_A);
    free(h_B);
    free(h_C);

    printf("Multiplicación completada correctamente.\n");
    return 0;
}
